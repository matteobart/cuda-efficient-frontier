
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
//to get data yahoo finance
//time period: Apr 01 2016 -> Apr 01 2019
//freq: Weekly
#define NUM_ELEMENTS 100 //why when I change this everything breaks & this should not change because constant memory 
#define NUM_PORTFOLIOS atoi(argv[argc])
#define MAX_NUM_OF_STOCKS 85

#define DEBUG 0

float* readFile(char* filename){
    float* ret = (float*) malloc(NUM_ELEMENTS*sizeof(float));
    FILE* ptr = fopen(filename,"r");
    if (ptr==NULL) 
    { 
        printf("Error reading file"); 
        return 0; 
    } 
    char line[255];
    char* token;
    int lineCount = 0; 
    fgets(line, 255, ptr); //grab the first line and do nothing
    while (fgets(line, 255, ptr) != 0 && lineCount < NUM_ELEMENTS){ //for each line
        int dataCount = 0;
        token = strtok(line, ",");
        while (token != 0) { //for each word in line
            if (dataCount == 5) {
                ret[lineCount] = atof(token);
            }
            token = strtok(0, ",");
            dataCount++;
        }
        lineCount++;
    }
    fclose(ptr);
    return ret;
}

void writeFile(char* filename, float* returns, float* risk, int len){
    FILE* ptr = fopen(filename, "w");
    for (int a = 0; a < len; a++){
        fprintf(ptr, "%f %f\n", risk[a], returns[a]);
    }
    fclose(ptr);

}

float getAverage(float* nums, int len){
    float sum = 0;
    for (int a = 0; a < len; a++){
        sum += nums[a];
    }
    return sum/len;
}

float* getPercentReturns(float* nums, int len){
    float* ret = (float*) malloc(sizeof(float)*(len-1));
    for (int a = 0; a < len-1; a++){
        ret[a] = (nums[a+1]-nums[a])/nums[a]; 
    }
    return ret; 
}

//a few possible errors in here
//still need to plot
//why am I mallocing
void gold(int argc, char* argv[]){
    argc--;
    if (argc < 3) {
        printf("%s\n", "Expected more arguments");
        exit(0);
    } 

    float** closingPrices = (float**) malloc(sizeof(float*)*(argc-1));
    float** returns = (float**) malloc(sizeof(float*)*(argc-1));
    float* averages = (float*) malloc(sizeof(float)*(argc-1));
    for (int a = 1; a < argc; a++){
        closingPrices[a-1] = readFile(argv[a]);
        returns[a-1] = getPercentReturns(closingPrices[a-1], NUM_ELEMENTS);
        averages[a-1] = getAverage(returns[a-1], NUM_ELEMENTS-1);
    }

    if (DEBUG){
        for (int a = 0; a < (argc-1); a++){
            for (int b = 0; b < (NUM_ELEMENTS-1); b++){
                printf("Returns %d %d: %f \n", a, b, returns[a][b]);
            }
        }

        for (int a = 0; a < argc-1; a++){
            printf("avg %d: %f\n", a, averages[a]);
        }
    }

    //calculate the covariances for each of the stocks 
    //doing extra things [0][4] will be the same as [4][0]
    float** covariance = (float**) malloc(sizeof(float*)*(argc-1));
    for (int a = 0; a < argc-1; a++){
        covariance[a] = (float*) malloc(sizeof(float)*(argc-1));
        for (int b = 0; b < argc-1; b++){
            float sum = 0;
            for (int c = 0; c < NUM_ELEMENTS-1; c++){
                sum += (returns[a][c] - averages[a]) * (returns[b][c] - averages[b]);
            }
            sum /= NUM_ELEMENTS-2;
            covariance[a][b] = sum;
        }
    }
    //retiming -malloc, transfers -constant *
    //transpose data array vs constant *
    //cutting down on local variables in portfolios
    //remove std all together *
    //move all the constants to pt2
    //dont double calculate for the covariance *
    //where do you put the write file *

    //time to choose the weights for the given portfolios
    //PSUDEO:
        //for doing random weights
        //if x stocks 
        //then choose x numbers
        //then find the sum of the randoms
        //then divide each random number by sum
    clock_t start = clock(), diff;
   
    srand(time(NULL));   // Initialization, should only be called once.
    float* risk = (float*) malloc(sizeof(float)* NUM_PORTFOLIOS);
    float* reward = (float*) malloc(sizeof(float)* NUM_PORTFOLIOS);
    for (int a = 0; a < NUM_PORTFOLIOS; a++){//find the risk & reward for each portfolio
        float randomWeights[argc-1]; //may actually want to save this for later
        int totalWeight = 0;
        for (int b = 0; b < argc-1; b++){//choose random weights
            int r = rand() % 100;  //RAND MIGHT BE DOING THE SAME VAL EVERYTIME
            totalWeight += r;  
            randomWeights[b] = (float) r;
        }
        for (int b = 0; b < argc-1; b++){//now random weight has the correct weights
            randomWeights[b] /= totalWeight;
        }

        //first find the reward
        float totalReward = 0;
        for (int b = 0; b < argc-1; b++){
            totalReward += averages[b]*randomWeights[b];

        }
        reward[a] = totalReward;

        //find the risk of the portfolio
        float totalRisk = 0;
        float work[argc-1];
        for (int b = 0; b < argc-1; b++){
            work[b] = 0;
            for (int c = 0; c < argc-1; c++){
                work[b] += randomWeights[c]*covariance[c][b];
            }
        }
        for (int b = 0; b < argc-1; b++){
            totalRisk += work[b] * randomWeights[b];
        }

        risk[a] = sqrt(totalRisk);
        if (a==0 && DEBUG){
            for (int r = 0; r < argc-1;r++) printf("randomWeights: %f\n", randomWeights[r]);
            printf("Risk: %f\n", risk[a]);
            for (int r = 0; r < argc-1; r++){
                for (int rr = 0; rr < argc-1; rr++){
                    printf("Cov of %d %d : %f\n", r, rr, covariance[r][rr]);
                }
            }
        } 
    }

    diff = clock() - start;
    float msec = (float) diff * 1000 / (float) CLOCKS_PER_SEC;
    printf("CPU time portfolio %f seconds\n", msec/1000);


    //plot the data
    if (DEBUG) writeFile("riskreturngold.txt", reward, risk, NUM_PORTFOLIOS);

}

__constant__ float c_returns[MAX_NUM_OF_STOCKS * 99];
__constant__ float c_averages[MAX_NUM_OF_STOCKS];
__constant__ float c_covariance[MAX_NUM_OF_STOCKS*MAX_NUM_OF_STOCKS];

__global__ void GPercentReturns(float* closingPrices, float* returns, int numOfStocks)
{
    __shared__ float closing[NUM_ELEMENTS];
    int stockId = blockIdx.x;
    int returnId = threadIdx.x; 

    int grab = returnId + (stockId * NUM_ELEMENTS); //also write 2

    //everyone load into shared
    closing[returnId] = closingPrices[grab];
    __syncthreads();

    if (returnId != NUM_ELEMENTS-1){//last thread should do this
        int to = returnId + (stockId*(NUM_ELEMENTS-1));
        returns[to] = (closing[returnId+1]-closing[returnId])/closing[returnId];

    }
}
__global__ void GReduceAverageR(float* average, int numOfStocks, int mid){
    __shared__ float reduce[99];

    int returnId = threadIdx.x;
    int stockId = blockIdx.x;
    int dim = blockDim.x;


    reduce[returnId] = c_returns[returnId + (stockId*dim)];
    __syncthreads();

    if (returnId>=mid){
        reduce[returnId-mid]+=reduce[returnId];
    }
    __syncthreads();

    for (int s = mid/2; s > 0; s/=2){
        if (returnId < s) {
            reduce[returnId] += reduce[returnId+s];
        }
        __syncthreads();
    }

    //REMOVED IF
    if (returnId == 0) average[stockId] = reduce[0]/99.0;
    
}

//try flipping the memory 
__global__ void GCovariance(float* covariance, int numberOfStocks){
    int b = threadIdx.x;
    int a = blockIdx.x;

    if (a > b) return;

    float sum = 0;
    for (int c = 0; c < NUM_ELEMENTS-1; c++)
        sum += (c_returns[a*(NUM_ELEMENTS-1)+c] - c_averages[a]) * (c_returns[b*(NUM_ELEMENTS-1)+c] - c_averages[b]);
    
    sum /= NUM_ELEMENTS-2;
    covariance[a*numberOfStocks+b] = sum;
    covariance[b*numberOfStocks+a] = sum;
}


__global__ void GPortfolio(float* risk, float* reward, int numberOfStocks, int mid){
    //obscene amount of global calls here
    //only one call to risk[] and reward[] at the end
    //also there might be a GPU version of sqrt()
    extern __shared__ float sharedMemory[];
    float* randomWeights = (float*) &sharedMemory[0];
    float* scratch = (float*) &sharedMemory[numberOfStocks];
    
    //__shared__ float randomWeights[16];
    //__shared__ float scratch[16];


    int tid = threadIdx.x;
    int bid = blockIdx.x;

    hiprandState state;
    hiprand_init(tid+bid*blockDim.x,10,0,&state);
    float r = hiprand_uniform(&state);
    
    //RAN WEIGHT
    //FAST- WORKS
    randomWeights[tid] = r;
    __syncthreads();
    //quick reduce
    if (tid >= mid){
        randomWeights[tid-mid] += randomWeights[tid];
    }
    __syncthreads();

    for (int s = mid/2; s > 0; s /= 2){
        if (tid < s) 
            randomWeights[tid] += randomWeights[tid+s];
        __syncthreads();
    }
    float totalWeight = randomWeights[0];
    __syncthreads();
    randomWeights[tid] = (float) r/ totalWeight;

    //RETURN
    //FAST


    scratch[tid] = c_averages[tid]*randomWeights[tid];
    __syncthreads();
    if (tid >= mid){
        scratch[tid-mid] += scratch[tid];
        if (tid >= numberOfStocks) printf("%d\n", tid);
        if (tid-mid < 0) printf("%d", tid-mid);
    }
    __syncthreads();
    for (int s = mid/2; s > 0; s /= 2){
        if (tid < s) {
            scratch[tid] += scratch[tid+s];

        }
        __syncthreads();
    }
    if (tid == 0) reward[bid] = scratch[0];
    __syncthreads();


    //RISK
    //FAST
    float work = 0;
    for (int c = 0; c < numberOfStocks; c++){
         work += randomWeights[c]*c_covariance[c*numberOfStocks+tid];
    }
    scratch[tid] = work*randomWeights[tid];

    __syncthreads();
    if (tid >= mid){
        scratch[tid-mid] += scratch[tid];
    }
    __syncthreads();

    for (int s = mid/2; s > 0; s /= 2){
        if (tid < s) 
            scratch[tid] += scratch[tid+s];
        __syncthreads();
    }

    if (tid == 0) risk[bid] = sqrt(scratch[0]);

}

void gpu (int argc, char* argv[]) {
    argc--;
    float* closingPrices = (float*) malloc(sizeof(float)*(argc-1)*NUM_ELEMENTS);
    float* returns = (float*) malloc(sizeof(float)*(argc-1)*(NUM_ELEMENTS-1));
    float* averages = (float*) malloc(sizeof(float)*(argc-1));
    float* std = (float*) malloc(sizeof(float)*(argc-1));
    float* covariance = (float*) malloc(sizeof(float)*(argc-1)*(argc-1));

    for (int a = 1; a < argc; a++){
        float* add = readFile(argv[a]);
        for (int b = 0; b < NUM_ELEMENTS; b++){
            closingPrices[(a-1)*NUM_ELEMENTS+b] = add[b];
        }
    }
    float* d_closingPrices;
    hipMalloc(&d_closingPrices, sizeof(float) * (argc-1)*NUM_ELEMENTS);

    float* d_all;
    hipMalloc(&d_all, sizeof(float) * (argc-1)*(NUM_ELEMENTS-1));


    hipMemcpy(d_closingPrices, closingPrices, sizeof(float)*(argc-1)*NUM_ELEMENTS, hipMemcpyHostToDevice);


    GPercentReturns<<<argc-1,NUM_ELEMENTS>>>(d_closingPrices, d_all, argc-1);
    hipMemcpy(returns, d_all, sizeof(float)*(argc-1)*(NUM_ELEMENTS-1), hipMemcpyDeviceToHost);
    hipMemcpyToSymbol(HIP_SYMBOL(c_returns), returns, sizeof(float) * (argc-1)*(NUM_ELEMENTS-1));

    if (DEBUG){
        for (int a = 0; a < (argc-1); a++){
            for (int b = 0; b < (NUM_ELEMENTS-1); b++){
                printf("Returns %d %d: %f \n", a, b, returns[a*(NUM_ELEMENTS-1)+b]);
            }
        }
    }

    int mid = 1;
    while (mid * 2 <= NUM_ELEMENTS-1) {
        mid *= 2;
    }

    GReduceAverageR<<<argc-1, NUM_ELEMENTS-1>>>(d_all, argc-1, mid);

    hipMemcpy(averages, d_all, sizeof(float)*(argc-1), hipMemcpyDeviceToHost);
    hipMemcpyToSymbol(HIP_SYMBOL(c_averages), averages, sizeof(float) * (argc-1));

    if (DEBUG){
        for (int a = 0; a < argc-1; a++){
            printf("avg %d: %f\n", a, averages[a]);
        }
    }   
    
    GCovariance<<<argc-1,argc-1>>>(d_all, argc-1);

    hipMemcpy(covariance, d_all, sizeof(float)*(argc-1)*(argc-1), hipMemcpyDeviceToHost);
    hipMemcpyToSymbol(HIP_SYMBOL(c_covariance), covariance, sizeof(float) * (argc-1)*(argc-1));

    if (DEBUG){
        for (int a = 0; a < argc-1; a++){
            for (int b = 0; b <argc-1; b++){
                printf("Cov %d %d: %f\n", a, b, covariance[a*(argc-1)+b]);
            }
        }
    }

    //timing just for portfolio
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    //END

    float* risk = (float*) malloc(sizeof(float)*NUM_PORTFOLIOS);
    float* reward = (float*) malloc(sizeof(float)*NUM_PORTFOLIOS);
    float* d_risk;
    float* d_reward;
    hipMalloc(&d_risk, sizeof(float)*NUM_PORTFOLIOS);
    hipMalloc(&d_reward, sizeof(float)*NUM_PORTFOLIOS);

    mid = 1;
    while (mid * 2 <= argc-1){
        mid *= 2;
    }
    GPortfolio<<<NUM_PORTFOLIOS, argc-1, (sizeof(float)*(argc-1))*2>>>(d_risk, d_reward, argc-1, mid);
    hipMemcpy(risk, d_risk, sizeof(float)*NUM_PORTFOLIOS, hipMemcpyDeviceToHost);
    hipMemcpy(reward, d_reward, sizeof(float)*NUM_PORTFOLIOS, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    //START
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipError_t code = hipEventElapsedTime(&time, start, stop);
    if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
      
    }
    //END
    printf("Time for portfolio: %f s\n", time/1000);

    if (DEBUG) writeFile("riskreturn.txt", reward, risk, NUM_PORTFOLIOS);

}
//to plot
//in terminal do 
//gnuplot
//plot 'riskreturn.txt' with points pt 3



int main( int argc, char* argv[])
{
    printf("Num stocks: %i \n", argc-2);
    printf("Num port: %i \n", atoi(argv[argc-1]));

    clock_t start = clock(), diff;
    gold(argc, argv);
    diff = clock() - start;
    float msec = (float) diff * 1000 / (float) CLOCKS_PER_SEC;
    printf("Total CPU time taken %f seconds\n", msec/1000);
    

    clock_t start2 = clock(), diff2;
    gpu(argc, argv);
    diff2 = clock() - start2;
    float msec2 = (float) diff2 * 1000 / (float) CLOCKS_PER_SEC;
    printf("Total GPU time taken %f seconds \n", msec2/1000);


    return 0;
}