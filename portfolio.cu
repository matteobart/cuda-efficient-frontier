
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
//to get data yahoo finance
//time period: Apr 01 2016 -> Apr 01 2019
//freq: Weekly
#define NUM_ELEMENTS 100 //why when I change this everything breaks
#define NUM_PORTFOLIOS atoi(argv[argc])
#define MAX_NUM_OF_STOCKS 158


float* readFile(char* filename){
    float* ret = (float*) malloc(NUM_ELEMENTS*sizeof(float));
    FILE* ptr = fopen(filename,"r");
    if (ptr==NULL) 
    { 
        printf("Error reading file"); 
        return 0; 
    } 
    char line[255];
    char* token;
    int lineCount = 0; 
    fgets(line, 255, ptr); //grab the first line and do nothing
    while (fgets(line, 255, ptr) != 0 && lineCount < NUM_ELEMENTS){ //for each line
        int dataCount = 0;
        token = strtok(line, ",");
        while (token != 0) { //for each word in line
            if (dataCount == 5) {
                ret[lineCount] = atof(token);
            }
            token = strtok(0, ",");
            dataCount++;
        }
        lineCount++;
    }
    fclose(ptr);
    return ret;
}

void writeFile(char* filename, float* returns, float* risk, int len){
    FILE* ptr = fopen(filename, "w");
    for (int a = 0; a < len; a++){
        fprintf(ptr, "%f %f\n", risk[a], returns[a]);
    }
    fclose(ptr);

}

float getAverage(float* nums, int len){
    float sum = 0;
    for (int a = 0; a < len; a++){
        sum += nums[a];
    }
    return sum/len;
}

float* getPercentReturns(float* nums, int len){
    float* ret = (float*) malloc(sizeof(float)*(len-1));
    for (int a = 0; a < len-1; a++){
        ret[a] = (nums[a+1]-nums[a])/nums[a]; 
    }
    return ret; 
}

//a few possible errors in here
//still need to plot
//why am I mallocing
void gold(int argc, char* argv[]){
    argc--;
    if (argc < 3) {
        printf("%s\n", "Expected more arguments");
        exit(0);
    } 

    float** closingPrices = (float**) malloc(sizeof(float*)*(argc-1));
    float** returns = (float**) malloc(sizeof(float*)*(argc-1));
    float* averages = (float*) malloc(sizeof(float)*(argc-1));

    for (int a = 1; a < argc; a++){
        closingPrices[a-1] = readFile(argv[a]);
        returns[a-1] = getPercentReturns(closingPrices[a-1], NUM_ELEMENTS);
        averages[a-1] = getAverage(returns[a-1], NUM_ELEMENTS-1);
    }

    for (int a = 0; a < (argc-1); a++){
        for (int b = 0; b < (NUM_ELEMENTS-1); b++){
            printf("Returns %d %d: %f \n", a, b, returns[a][b]);
        }
    }

    for (int a = 0; a < argc-1; a++){
        printf("avg %d: %f\n", a, averages[a]);
    }

    //calculate the standard deviation
    float* std = (float*) malloc(sizeof(float)*(argc-1));
    for (int a = 0; a < argc-1; a++){
        for (int b = 0; b < NUM_ELEMENTS-1; b++){
            std[a] += pow(returns[a][b]-averages[a], 2);   
        }
        std[a] /= NUM_ELEMENTS-2;
        std[a] = sqrt(std[a]);
    }

    for (int a = 0; a < argc-1; a++){
        printf("Std %d: %f \n", a, std[a]);
    }

    //calculate the covariances for each of the stocks 
    //doing extra things [0][4] will be the same as [4][0]
    float** covariance = (float**) malloc(sizeof(float*)*(argc-1));
    for (int a = 0; a < argc-1; a++){
        covariance[a] = (float*) malloc(sizeof(float)*(argc-1));
        for (int b = 0; b < argc-1; b++){
            float sum = 0;
            for (int c = 0; c < NUM_ELEMENTS-1; c++){
                sum += (returns[a][c] - averages[a]) * (returns[b][c] - averages[b]);
            }
            sum /= NUM_ELEMENTS-2;
            covariance[a][b] = sum;
            printf("%f\n", sum);
        }
    }

    //time to choose the weights for the given portfolios
    //PSUDEO:
        //for doing random weights
        //if x stocks 
        //then choose x numbers
        //then find the sum of the randoms
        //then divide each random number by sum
    clock_t start = clock(), diff;
   
    srand(time(NULL));   // Initialization, should only be called once.
    float* risk = (float*) malloc(sizeof(float)* NUM_PORTFOLIOS);
    float* reward =(float*) malloc(sizeof(float)* NUM_PORTFOLIOS);
    for (int a = 0; a < NUM_PORTFOLIOS; a++){//find the risk & reward for each portfolio
        float randomWeights[argc-1]; //may actually want to save this for later
        int totalWeight = 0;
        for (int b = 0; b < argc-1; b++){//choose random weights
            int r = rand() % 100;  //RAND MIGHT BE DOING THE SAME VAL EVERYTIME
            totalWeight += r;  
            randomWeights[b] = (float) r;
        }
        for (int b = 0; b < argc-1; b++){//now random weight has the correct weights
            randomWeights[b] /= totalWeight;
        }

        //first find the reward
        float totalReward = 0;
        for (int b = 0; b < argc-1; b++){
            totalReward += averages[b]*randomWeights[b];

        }
        reward[a] = totalReward;

        //find the risk of the portfolio
        float totalRisk = 0;
        float work[argc-1];
        for (int b = 0; b < argc-1; b++){
            work[b] = 0;
            for (int c = 0; c < argc-1; c++){
                work[b] += randomWeights[c]*covariance[c][b];
            }
        }
        for (int b = 0; b < argc-1; b++){
            totalRisk += work[b] * randomWeights[b];
        }

        risk[a] = sqrt(totalRisk);
        if (a==0){
            for (int r = 0; r < argc-1;r++) printf("randomWeights: %f\n", randomWeights[r]);
            printf("Risk: %f\n", risk[a]);
            for (int r = 0; r < argc-1; r++){
                for (int rr = 0; rr < argc-1; rr++){
                    printf("Cov of %d %d : %f\n", r, rr, covariance[r][rr]);
                }
            }
        } 
    }

    diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("Time taken for just portfolios %d seconds %d milliseconds\n", msec/1000, msec%1000);
    

    //plot the data
    writeFile("riskreturngold.txt", reward, risk, NUM_PORTFOLIOS);

}

__constant__ float c_returns[MAX_NUM_OF_STOCKS * 99];
__constant__ float c_averages[MAX_NUM_OF_STOCKS];
__constant__ float c_std[MAX_NUM_OF_STOCKS];

__global__ void GPercentReturns(float* closingPrices, float* returns, int numOfStocks)
{
    __shared__ float closing[NUM_ELEMENTS];
    int stockId = blockIdx.x;
    int returnId = threadIdx.x; 

    int grab = returnId + (stockId * NUM_ELEMENTS); //also write 2

    //everyone load into shared
    closing[returnId] = closingPrices[grab];
    __syncthreads();

    if (returnId != NUM_ELEMENTS-1){//last thread should do this
        int to = returnId + (stockId*(NUM_ELEMENTS-1));
        returns[to] = (closing[returnId+1]-closing[returnId])/closing[returnId];

    }
    //int grab2 = returnId+1 + (stockId * NUM_ELEMENTS); 

    //returns[to] = (closingPrices[grab2]-closingPrices[grab1])/closingPrices[grab1];
}

//where mid is greater power of 2 less than or equal to number of elements
__global__ void GReduceAverage(float* average, int numOfStocks, int mid){
    int returnId = threadIdx.x;
    //int stockId = blockIdx.x;
    //int dim = blockDim.x;

    float tot = 0; 
    for(int a = 0; a < NUM_ELEMENTS-1; a++){
        tot += c_returns[a+(returnId*(NUM_ELEMENTS-1))];
    }
    average[returnId] = tot / (float) (NUM_ELEMENTS-1);
    /*
    if (returnId<mid && returnId+mid<dim){
        returns[(stockId*dim)+returnId]+=returns[(stockId*dim)+returnId+mid];
    }
    for (int s = mid/2; s > 0; s/=2){
        if (returnId < s) {
            returns[(stockId*dim)+returnId] += returns[(stockId*dim)+returnId+s];
        }
    }
    if(returnId == 0){
        average[stockId] = returns[stockId*dim]/dim;
    }*/
    
}


__global__ void GStd(float* std, int numOfStocks, int mid){
    extern __shared__ float s_std[];
    int returnId = threadIdx.x;
    int stockId = blockIdx.x;
    int dim = blockDim.x;
    float add = powf(c_returns[stockId*(NUM_ELEMENTS-1)+returnId]-c_averages[stockId], 2);
    s_std[returnId] = add;
    __syncthreads();

    //if (returnId<mid && returnId+mid<dim){
    //    s_std[returnId]+=s_std[returnId+mid];
    //}

    if (returnId>=mid){
        s_std[returnId-mid]+=s_std[returnId];
    }
    __syncthreads();
    for (int s = mid/2; s > 0; s/=2){
        if (returnId < s) {
            s_std[returnId]+=s_std[returnId+s];
        }
        __syncthreads();
    }


    //atomicAdd(&std[stockId], powf(returns[stockId*(NUM_ELEMENTS-1)+returnId]-averages[stockId], 2));
    __syncthreads();
    if (returnId == 0) {
        std[stockId]= sqrt(s_std[0]/(NUM_ELEMENTS-2));
    }
}   

__global__ void GCovariance(float* covariance, int numberOfStocks){
   // __shared__ float s_returns[]

    int b = threadIdx.x;
    int a = threadIdx.y;

    float sum = 0;
    for (int c = 0; c < NUM_ELEMENTS-1; c++)
        sum += (c_returns[a*(NUM_ELEMENTS-1)+c] - c_averages[a]) * (c_returns[b*(NUM_ELEMENTS-1)+c] - c_averages[b]);
    
    sum /= NUM_ELEMENTS-2;
    covariance[a*numberOfStocks+b] = sum;
}


__global__ void GPortfolio(hiprandState*state, float* covariance, float* risk, float* reward, int numberOfStocks, int mid){
    //obscene amount of global calls here
    //only one call to risk[] and reward[] at the end
    //also there might be a GPU version of sqrt()
    extern __shared__ float sharedMemory[];
    float* randomWeights = (float*) &sharedMemory[0];
    float* scratch = (float*) &sharedMemory[numberOfStocks];
    
    //__shared__ float randomWeights[16];
    //__shared__ float scratch[16];


    int tid = threadIdx.x;
    int bid = blockIdx.x;

    float r = hiprand_uniform(&state[tid+bid*blockDim.x]);
    
    //RAN WEIGHT
    //FAST- WORKS
    randomWeights[tid] = r;
    __syncthreads();
    //quick reduce
    if (tid >= mid){
    //if (tid<mid && tid+mid<numberOfStocks){
        //randomWeights[tid] += randomWeights[tid+mid];

        randomWeights[tid-mid] += randomWeights[tid];
    }
    __syncthreads();

    for (int s = mid/2; s > 0; s /= 2){
        if (tid < s) 
            randomWeights[tid] += randomWeights[tid+s];
        __syncthreads();
    }
    float totalWeight = randomWeights[0];
    __syncthreads();
    randomWeights[tid] = (float) r/ totalWeight;

    //SLOW-IGNORE
    // randomWeights[tid] = r;
    // __syncthreads();
    // atomicAdd(&randomWeights[0], r);
    // __syncthreads();
    // float totalWeight = randomWeights[0];
    // __syncthreads();
    // randomWeights[tid] = r/totalWeight;

    //RETURN
    //FAST


    scratch[tid] = c_averages[tid]*randomWeights[tid];
    __syncthreads();
    if (tid >= mid){
        scratch[tid-mid] += scratch[tid];
        if (tid >= numberOfStocks) printf("%d\n", tid);
        if (tid-mid < 0) printf("%d", tid-mid);
    }
    __syncthreads();
    for (int s = mid/2; s > 0; s /= 2){
        if (tid < s) {
            scratch[tid] += scratch[tid+s];
            //printf("i %d %d\n",tid ,tid+s);

        }
        __syncthreads();
    }
    reward[bid] = scratch[0];
    __syncthreads();

    //SLOW
    // reward[bid] = 0;
    // atomicAdd(&reward[bid], averages[tid]*randomWeights[tid]);
    // __syncthreads();
    //if (tid == 0) printf("%d: %f %f\n", bid, reward[bid], deleteMe);

    //RISK
    //FAST
    float work = 0;
    for (int c = 0; c < numberOfStocks; c++){
         work += randomWeights[c]*covariance[c*numberOfStocks+tid];
    }
    scratch[tid] = work*randomWeights[tid];

    __syncthreads();
    if (tid >= mid){
        scratch[tid-mid] += scratch[tid];
    }
    __syncthreads();

    for (int s = mid/2; s > 0; s /= 2){
        if (tid < s) 
            scratch[tid] += scratch[tid+s];
        __syncthreads();
    }
    risk[bid] = sqrt(scratch[0]);

    //SLOW
    // float work = 0;
    // for (int c = 0; c < numberOfStocks; c++){
    //      work += randomWeights[c]*covariance[c*numberOfStocks+tid];
    // }

    // atomicAdd(&risk[bid], work * randomWeights[tid]);
    // __syncthreads();
    // if (tid == 0)
    //     risk[bid] = sqrt(risk[bid]);
}

__global__ void init_stuff(hiprandState*state){int idx=blockIdx.x*blockDim.x+threadIdx.x;hiprand_init(1337,idx,0,&state[idx]);}


void gpu (int argc, char* argv[]) {
    argc--;
    float* closingPrices = (float*) malloc(sizeof(float)*(argc-1)*NUM_ELEMENTS);
    float* returns = (float*) malloc(sizeof(float)*(argc-1)*(NUM_ELEMENTS-1));
    float* averages = (float*) malloc(sizeof(float)*(argc-1));

    for (int a = 1; a < argc; a++){
        float* add = readFile(argv[a]);
        for (int b = 0; b < NUM_ELEMENTS; b++){
            closingPrices[(a-1)*NUM_ELEMENTS+b] = add[b];
        }
    }
    float* d_closingPrices;
    hipMalloc(&d_closingPrices, sizeof(float) * (argc-1)*NUM_ELEMENTS);

    float* d_returns;
    hipMalloc(&d_returns, sizeof(float) * (argc-1)*(NUM_ELEMENTS-1));


    float* d_averages;
    hipMalloc(&d_averages, sizeof(float) * (argc-1));

    hipMemcpy(d_closingPrices, closingPrices, sizeof(float)*(argc-1)*NUM_ELEMENTS, hipMemcpyHostToDevice);


    dim3 blockSize (NUM_ELEMENTS-1, argc-1);
    GPercentReturns<<<argc-1,NUM_ELEMENTS>>>(d_closingPrices, d_returns, argc-1);
    hipMemcpy(returns, d_returns, sizeof(float)*(argc-1)*(NUM_ELEMENTS-1), hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); //is this needed here
    hipMemcpyToSymbol(HIP_SYMBOL(c_returns), returns, sizeof(float) * (argc-1)*(NUM_ELEMENTS-1));

    for (int a = 0; a < (argc-1); a++){
        for (int b = 0; b < (NUM_ELEMENTS-1); b++){
            printf("Returns %d %d: %f \n", a, b, returns[a*(NUM_ELEMENTS-1)+b]);
        }
    }

    float* d_work; //should be init with returns
    hipMalloc(&d_work, sizeof(float)*(argc-1)*(NUM_ELEMENTS-1));
    hipMemcpy(d_work, d_returns, sizeof(float)*(argc-1)*(NUM_ELEMENTS-1),hipMemcpyDeviceToDevice);
    //NUM_ELEMENTS/2 because its reduce... may change
    int mid = 1;
    while (mid * 2 <= NUM_ELEMENTS-1) {
        mid *= 2;
    }
    //smater
    //GReduceAverage<<<argc-1, NUM_ELEMENTS/2>>>(d_work, d_averages, argc-1, mid);
    //dumber
    GReduceAverage<<<1, argc-1>>>(d_averages, argc-1, mid);
    hipMemcpy(averages, d_averages, sizeof(float)*(argc-1), hipMemcpyDeviceToHost);
    hipMemcpyToSymbol(HIP_SYMBOL(c_averages), averages, sizeof(float) * (argc-1));

    for (int a = 0; a < argc-1; a++){
        printf("avg %d: %f\n", a, averages[a]);
    }

    float* std = (float*) malloc(sizeof(float)*(argc-1));

    float* d_std;
    hipMalloc(&d_std, sizeof(float)*(argc-1)); 

    GStd<<<argc-1, NUM_ELEMENTS-1, sizeof(float)*(NUM_ELEMENTS-1)>>>(d_std, argc-1, mid);

    hipMemcpy(std, d_std, sizeof(float)*(argc-1), hipMemcpyDeviceToHost);
    hipMemcpyToSymbol(HIP_SYMBOL(c_std), std, sizeof(float) * (argc-1));

    for (int a = 0; a < argc-1; a++){
        printf("Std %d: %f \n", a, std[a]);
    }

    float* covariance = (float*) malloc(sizeof(float)*(argc-1)*(argc-1));//2D like array
    float* d_covariance;
    hipMalloc(&d_covariance, sizeof(float)*(argc-1)*(argc-1));
    
    blockSize.x = argc-1;
    blockSize.y = argc-1;
    GCovariance<<<1,blockSize>>>(d_covariance, argc-1);
    hipMemcpy(covariance, d_covariance, sizeof(float)*(argc-1)*(argc-1), hipMemcpyDeviceToHost);

    for (int a = 0; a < argc-1; a++){
        for (int b = 0; b <argc-1; b++){
            printf("Cov %d %d: %f\n", a, b, covariance[a*(argc-1)+b]);
        }
    }

    //timing just for portfolio
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    //END

    float* risk = (float*) malloc(sizeof(float)*NUM_PORTFOLIOS);
    float* reward = (float*) malloc(sizeof(float)*NUM_PORTFOLIOS);
    float* d_risk;
    float* d_reward;
    hipMalloc(&d_risk, sizeof(float)*NUM_PORTFOLIOS);
    hipMalloc(&d_reward, sizeof(float)*NUM_PORTFOLIOS);

    hiprandState*d_state;
    hipMalloc(&d_state,(argc-1)*NUM_PORTFOLIOS);
    init_stuff<<<NUM_PORTFOLIOS,argc-1>>>(d_state);

    mid = 1;
    while (mid * 2 <= argc-1){
        mid *= 2;
    }
    GPortfolio<<<NUM_PORTFOLIOS, argc-1, (sizeof(float)*(argc-1))*2>>>(d_state, d_covariance, d_risk, d_reward, argc-1, mid);
    printf("Middy %d\n",mid);
    hipMemcpy(risk, d_risk, sizeof(float)*NUM_PORTFOLIOS, hipMemcpyDeviceToHost);
    hipMemcpy(reward, d_reward, sizeof(float)*NUM_PORTFOLIOS, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    //START
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipError_t code = hipEventElapsedTime(&time, start, stop);
    if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
      
    }
    //END
    printf("Time for portfolio: %f s\n", time/1000);

    writeFile("riskreturn.txt", reward, risk, NUM_PORTFOLIOS);

}
//to plot
//in terminal do 
//gnuplot
//plot 'riskreturn.txt' with points pt 3



int main( int argc, char* argv[])
{

    clock_t start = clock(), diff;
    gold(argc, argv);
    diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("Time taken %d seconds %d milliseconds\n", msec/1000, msec%1000);
    

    clock_t start2 = clock(), diff2;
    gpu(argc, argv);
    diff2 = clock() - start2;
    int msec2 = diff2 * 1000 / CLOCKS_PER_SEC;
    printf("Time taken %d seconds %d milliseconds\n", msec2/1000, msec2%1000);


    return 0;
}